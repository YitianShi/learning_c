#include<iostream>
#include<algorithm>
#include<vector>
#include<hip/hip_runtime.h>
#include <cstdlib>
using namespace std;
const int BLOCK= 25;
__global__
void vectAdd(float* A, float* B, float* C, int n)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    if (i<n) C[i] = A[i]+B[i];
}

__global__
void vectMul(float* A, float* B, float* C, int M, int N, int K)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int j = threadIdx.y + blockDim.y*blockIdx.y;
    if (i < N && j < M)
    {
    float sum=0;
    for (size_t k=0; k<K; k++)
    {
        sum += A[j*K + k] * A[k*N + i];
    }
    C[j*N + i] = sum;
    };
}

__global__
void vectMul2(float* A, float* B, float* C, int width, int numARow, int numBCol)
{
    __shared__ float sharedM[BLOCK][BLOCK];
    __shared__ float sharedN[BLOCK][BLOCK];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int r = by*BLOCK + ty;
    int c = bx*BLOCK + tx;

    float Csub = 0.0;
    for (int i = 0; i < (int)(ceil((float)width / BLOCK));i++)
    {
        if (i*BLOCK + tx < width && r < numARow)
        sharedM[ty][tx] = A[r*width + i*BLOCK + tx];
        else
        sharedM[ty][tx] = 0.0;

        if (i*BLOCK + ty < width && c < numBCol)
        sharedN[ty][tx] = B[(i*BLOCK + ty)*width + c];
        else
        sharedN[ty][tx] = 0.0;

        __syncthreads();

        for (int j = 0; j< BLOCK; j++)
        Csub += sharedM[ty][j] * sharedN[j][tx];
        __syncthreads();
    }

    if (r<numARow && c<numBCol)
    C[r*numBCol + c] = Csub;
}



int main(int argc, char *argv[])
{    
    int n = atoi(argv[1]);
    int size = n*sizeof(float);
    cout<<n<<endl;

    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C = (float *)malloc(size);

    for (int i = 0; i < n; i++) {
        float af = rand() / double(RAND_MAX);
        float bf = rand() / double(RAND_MAX);
        A[i] = af;
        B[i] = bf;
    }

    float* A_d, *B_d, *C_d, time_0;

    hipStream_t stream;
    hipEvent_t event, event2;
    hipStreamCreate(&stream);
    hipEventCreate(&event);
    hipEventCreate(&event2);

    hipMalloc((void**) &A_d, size);
    hipMalloc((void**) &B_d, size);
    hipMemcpyAsync(A_d, A, size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(B_d, B, size, hipMemcpyHostToDevice, stream);
    hipMalloc((void**) &C_d, size);
    hipMemcpyAsync(C, C_d, size, hipMemcpyDeviceToHost, stream);

    int thread_per_block = 256;
    int block_per_grid = (n + thread_per_block-1)/thread_per_block;

    dim3 DimGrid(block_per_grid, 1, 1);
    dim3 DimBlock(thread_per_block, 1, 1);

    //vectAdd <<<block_per_grid,thread_per_block, 0, stream>>>(A_d, B_d, C_d, n);
    hipEventRecord(event, stream);
    vectMul2 <<<block_per_grid,thread_per_block, 0, stream>>>(A_d, B_d, C_d, n, n, n);
    hipStreamSynchronize(stream);
    hipEventRecord(event2, stream);

    hipEventSynchronize(event);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&time_0, event, event2);


    hipStreamDestroy(stream);
    hipEventDestroy(event);
    hipEventDestroy(event2);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    printf("%f ms",time_0);
}



