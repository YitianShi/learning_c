#include<iostream>
#include<algorithm>
#include<vector>
#include<hip/hip_runtime.h>
#include <cstdlib>
using namespace std;
const int BLOCK= 25;
__global__
void vectAdd(float* A, float* B, float* C, int n)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    if (i<n) C[i] = A[i]+B[i];
}

__global__
void vectMul(float* A, float* B, float* C, int M, int N, int K)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int j = threadIdx.y + blockDim.y*blockIdx.y;
    if (i < N && j < M)
    {
    float sum=0;
    for (size_t k=0; k<K; k++)
    {
        sum += A[j*K + k] * A[k*N + i];
    }
    C[j*N + i] = sum;
    };
}

__global__
void vectMul2(float* A, float* B, float* C, int numACol, int numARow, int numBCol, int numBRow, int numCCol, int numCRow)
{
    __shared__ float sharedM[BLOCK][BLOCK];
    __shared__ float sharedN[BLOCK][BLOCK];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int r = by*BLOCK + ty;
    int c = bx*BLOCK + tx;

    float Csub = 0.0;
    for (int i = 0; i < (int)(ceil((float)numACol / BLOCK));i++)
    {
        if (i*BLOCK + tx < numACol && r < numARow)
        sharedM[ty][tx] = A[r*numACol + i*BLOCK + tx];
        else
        sharedM[ty][tx] = 0.0;

        if (i*BLOCK + ty < numBRow && c < numBCol)
        sharedN[ty][tx] = B[(i*BLOCK + ty)*numBCol + c];
        else
        sharedN[ty][tx] = 0.0;

        __syncthreads();

        for (int j = 0; j< BLOCK; j++)
        Csub += sharedM[ty][j] * sharedN[j][tx];
        __syncthreads();
    }

    if (r<numCRow && c<numCCol)
    C[r*numCCol + c] = Csub;
}



int main(int argc, char *argv[])
{    
    int n = atoi(argv[1]);
    int size = n*sizeof(float);
    cout<<n<<endl;

    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C = (float *)malloc(size);

    for (int i = 0; i < n; i++) {
        float af = rand() / double(RAND_MAX);
        float bf = rand() / double(RAND_MAX);
        A[i] = af;
        B[i] = bf;
    }

    float* A_d, *B_d, *C_d;

    hipMalloc((void**) &A_d, size);
    hipMalloc((void**) &B_d, size);
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &C_d, size);

    int thread_per_block = 256;
    int block_per_grid = (n + thread_per_block-1)/thread_per_block;

    dim3 DimGrid(block_per_grid, 1, 1);
    dim3 DimBlock(thread_per_block, 1, 1);

    vectAdd <<<block_per_grid,thread_per_block>>>(A_d, B_d, C_d, n);
    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}



