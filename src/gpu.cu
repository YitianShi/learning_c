#include "hip/hip_runtime.h"
#include<iostream>
#include<algorithm>
#include<vector>
#include<hip/hip_runtime.h>
#include <cstdlib>
#include <hipblas.h>
#include "NvInfer.h"
#include "NvInferPlugin.h"
#include "NvInferRuntimeCommon.h"
#include "NvInferRuntime.h"
#include "NvInferVersion.h"
#include "NvOnnxParser.h"
#include "NvInferPluginUtils.h"
#include "NvOnnxConfig.h"


using namespace std;
const int BLOCK= 25;
__global__
void vectAdd(float* A, float* B, float* C, int n)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    if (i<n) C[i] = A[i]+B[i];
}

__global__
void vectMul(float* A, float* B, float* C, int M, int N, int K)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int j = threadIdx.y + blockDim.y*blockIdx.y;
    if (i < N && j < M)
    {
    float sum=0;
    for (size_t k=0; k<K; k++)
    {
        sum += A[j*K + k] * A[k*N + i];
    }
    C[j*N + i] = sum;
    };
}

__global__
void vectMul2(float* A, float* B, float* C, int width, int numARow, int numBCol)
{
    __shared__ float sharedM[BLOCK][BLOCK];
    __shared__ float sharedN[BLOCK][BLOCK];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int r = by*BLOCK + ty;
    int c = bx*BLOCK + tx;

    float Csub = 0.0;
    for (int i = 0; i < (int)(ceil((float)width / BLOCK));i++)
    {
        if (i*BLOCK + tx < width && r < numARow)
        sharedM[ty][tx] = A[r*width + i*BLOCK + tx];
        else
        sharedM[ty][tx] = 0.0;

        if (i*BLOCK + ty < width && c < numBCol)
        sharedN[ty][tx] = B[(i*BLOCK + ty)*width + c];
        else
        sharedN[ty][tx] = 0.0;

        __syncthreads();

        for (int j = 0; j< BLOCK; j++)
        Csub += sharedM[ty][j] * sharedN[j][tx];
        __syncthreads();
    }

    if (r<numARow && c<numBCol)
    C[r*numBCol + c] = Csub;
}



int main(int argc, char *argv[])
{    
    int n = atoi(argv[1]);
    int size = n*sizeof(float);
    cout<<n<<endl;

    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C = (float *)malloc(size);

    for (int i = 0; i < n; i++) {
        float af = rand() / double(RAND_MAX);
        float bf = rand() / double(RAND_MAX);
        A[i] = af;
        B[i] = bf;
    }

    float* A_d, *B_d, *C_d, time_0;

    hipStream_t stream;
    hipEvent_t event, event2;
    hipStreamCreate(&stream);
    hipEventCreate(&event);
    hipEventCreate(&event2);

    hipMalloc((void**) &A_d, size);
    hipMalloc((void**) &B_d, size);
    hipMemcpyAsync(A_d, A, size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(B_d, B, size, hipMemcpyHostToDevice, stream);
    hipMalloc((void**) &C_d, size);
    hipMemcpyAsync(C, C_d, size, hipMemcpyDeviceToHost, stream);

    int thread_per_block = 256;
    int block_per_grid = (n + thread_per_block-1)/thread_per_block;

    dim3 DimGrid(block_per_grid, 1, 1);
    dim3 DimBlock(thread_per_block, 1, 1);

    //vectAdd <<<block_per_grid,thread_per_block, 0, stream>>>(A_d, B_d, C_d, n);
    hipEventRecord(event, stream);
    vectMul2 <<<block_per_grid,thread_per_block, 0, stream>>>(A_d, B_d, C_d, n, n, n);
    hipStreamSynchronize(stream);
    hipEventRecord(event2, stream);

    hipEventSynchronize(event);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&time_0, event, event2);

    hipStreamDestroy(stream);
    hipEventDestroy(event);
    hipEventDestroy(event2);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    printf("%f ms",time_0);
}

void cublas_try(unsigned int m, unsigned int n, unsigned int k,
                hipStream_t &streamId){

    //Initialize matrices in host
    float *h_A, *h_B, *h_C;

    unsigned int size_A = m*k;
    unsigned int mem_A = sizeof(float)*size_A;
    for (int i = 0; i < size_A; i++) {
        float af = rand() / double(RAND_MAX);
        h_A[i] = af;
    }

    unsigned int size_B = k*n;
    unsigned int mem_B = sizeof(float)*size_B;
    for (int i = 0; i < size_B; i++) {
        float af = rand() / double(RAND_MAX);
        h_B[i] = af;
    }

    unsigned int size_C = m*n;
    unsigned int mem_C = sizeof(float)*size_C;

    //Initialize matrices in device

    float *d_A, *d_B, *d_C;
    hipMalloc((void**) &d_A, mem_A);
    hipMalloc((void**) &d_B, mem_B);
    hipMalloc((void**) &d_C, mem_C);
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    
    dim3 DimGrid(1, 1);
    dim3 DimBlock(1, 1);

    float alpha=1.0f;
    float beta=0.0f;
    int batch = 25;
    long long int stride_A = 1, stride_B = 1, stride_C = 1;

    //create
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // hipblasStatus_t hipblasIsamax(handle, len, *x, row, *res); //hipblasIsamin
    
    /*  hipblasSgemv(
        handle, op,
        m, n, 
        &alpha,
        *A, &col_A (if op is HIPBLAS_OP_T else (HIPBLAS_OP_N) &row_A), 
        &x, &x_row, 
        &beta, 
        &y, y_row)
        
        alpha*op(A)*x + beta*y

        hipblasSgemm(
        handle, op_a, op_b, 
        m, n, k, 
        &alpha, 
        *A, &col_A (if op is HIPBLAS_OP_T else (HIPBLAS_OP_N) &row_A)
        *B, &col_B (if op is HIPBLAS_OP_T else (HIPBLAS_OP_N)&row_B),
        &beta, 
        *C, &row_C)

        C = alpha*op(A)*op(B) + beta*C
    */

    hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        m, n, k, 
        &alpha, 
        d_B, n, 
        d_A, k,
        &beta, 
        d_C, n);
    // C_T(n*m) = B_T (n*k) * A_T (k*m)

    hipblasSgemmBatched(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        m, n, k, 
        &alpha, 
        &d_B, n,
        &d_A, k,
        &beta, 
        &d_C, n, batch);
    //segmented sum
    
    hipblasSgemmStridedBatched(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        m, n, k, 
        &alpha, 
        d_B, n, stride_A,
        d_A, k, stride_B,
        &beta, 
        d_C, n, stride_C, 
        batch);
    // C+i*strideC = alpha*op(A+StrideA)*op(B+strIdeB) + beta*(C+i*strideC)

    
    hipDataType A_type = HIP_R_16F, B_type = HIP_R_16F, C_type = HIP_R_16F;
    hipblasComputeType_t Com_type = HIPBLAS_COMPUTE_16F;
    hipblasGemmEx(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        m, n, k, 
        &alpha, 
        d_B, B_type, n,
        d_A, A_type, k,
        &beta, 
        d_C, C_type, n, 
        Com_type, HIPBLAS_GEMM_DEFAULT);
    

    //stream
    /*
    hipblasSetStream(handle, streamId);
    hipblasGetStream(handle, &streamId);
    */

    

    //Calculate
    //hipblasSgemm()

    hipblasDestroy(handle);
    hipMemcpy(h_C, d_C, mem_C, hipMemcpyDeviceToHost);


}




